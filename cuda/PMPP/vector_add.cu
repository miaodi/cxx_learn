#include "hip/hip_runtime.h"
#include "vector_add.h"
__global__ void gpu_vector_add_kernel(const float *A, const float *B, float *C, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
    {
        C[idx] = A[idx] + B[idx];
    }
}

void gpu_vector_add(const float *A, const float *B, float *C, int N)
{
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    // copy to device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, N * sizeof(float));
    hipMalloc((void **)&d_B, N * sizeof(float));
    hipMalloc((void **)&d_C, N * sizeof(float));

    // copy to device memory
    hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);

    gpu_vector_add_kernel<<<numBlocks, blockSize>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    // copy back to host memory
    hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

void cpu_vector_add(const float *A, const float *B, float *C, int N)
{
#pragma omp parallel for
    for (int i = 0; i < N; ++i)
    {
        C[i] = A[i] + B[i];
    }
}