#include "hip/hip_runtime.h"
#include "gemm.h"
#include <hip/hip_runtime.h>
#include <stdexcept>

__global__ void gemm_kernel(const float *A, const float *B, float *C, int M,
                            int N, int K)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int row = idx / N;
    int col = idx % N;

    if (row < M && col < N)
    {
        float sum = 0.0f;
        for (int i = 0; i < K; ++i)
        {
            sum += A[row * K + i] * B[i * N + col];
        }
        C[row * N + col] = sum;
    }
}

void gpu_gemm(const float *A, const float *B, float *C, int M, int N, int K)
{
    int totalThreads = M * N;
    int blockSize = 256;
    int numBlocks = (totalThreads + blockSize - 1) / blockSize;

    float *d_A, *d_B, *d_C;
    size_t sizeA = M * K * sizeof(float);
    size_t sizeB = K * N * sizeof(float);
    size_t sizeC = M * N * sizeof(float);
    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);

    hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, sizeC, hipMemcpyHostToDevice);

    gemm_kernel<<<numBlocks, blockSize>>>(d_A, d_B, d_C, M, N, K);
    hipDeviceSynchronize();

    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

// Simple matrix transpose kernel (renamed from transpoe_kernel -> transpose_kernel)
__global__ void transpose_kernel(const float *input, float *output, int rows,
                                 int cols)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int row = idx / cols;
    int col = idx % cols;
    if (row < rows && col < cols)
    {
        output[col * rows + row] = input[row * cols + col];
    }
}

// // Host wrapper to launch transpose kernel (optional utility)
// void gpu_transpose(const float *input, float *output, int rows, int cols)
// {
//     int total = rows * cols;
//     int blockSize = 256;
//     int gridSize = (total + blockSize - 1) / blockSize;
//     transpose_kernel<<<gridSize, blockSize>>>(input, output, rows, cols);
//     hipDeviceSynchronize();
// }

void gpu_gemm_tiled(const float *A, const float *B, float *C, int M, int N,
                    int K, int tileSize)
{

    float *d_A, *d_B, *d_C;
    size_t sizeA = M * K * sizeof(float);
    size_t sizeB = K * N * sizeof(float);
    size_t sizeC = M * N * sizeof(float);
    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);
    hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, sizeC, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    int gridX = (N + tileSize - 1) / tileSize;
    int gridY = (M + tileSize - 1) / tileSize;

    dim3 gridDim(gridX, gridY);
    dim3 blockDim(tileSize, tileSize);

    switch (tileSize)
    {
    case 16:
        gemm_tiled_kernel<16><<<gridDim, blockDim>>>(d_A, d_B, d_C, M, N, K);
        break;
    case 32:
        gemm_tiled_kernel<32><<<gridDim, blockDim>>>(d_A, d_B, d_C, M, N, K);
        break;
    default:
        // Handle unsupported tile sizes
        throw std::invalid_argument("Unsupported tile size");
    }

    hipDeviceSynchronize();

    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

template <int TILE>
__global__ void gemm_tiled_kernel(
    const float *__restrict__ A, // M×K
    const float *__restrict__ B, // K×N
    float *__restrict__ C,       // M×N (output)
    int M, int N, int K)
{
    // Global row/col this thread computes in C
    int row = blockIdx.y * TILE + threadIdx.y;
    int col = blockIdx.x * TILE + threadIdx.x;

    // Shared tiles
    __shared__ float sA[TILE][TILE];
    __shared__ float sB[TILE][TILE];

    float acc = 0.0f;

    // Number of K-tiles
    int numTiles = (K + TILE - 1) / TILE;
    for (int t = 0; t < numTiles; ++t)
    {
        // Indices along K to load
        int aCol = t * TILE + threadIdx.x; // A column (K dimension)
        int bRow = t * TILE + threadIdx.y; // B row    (K dimension)

        // Cooperative loads (guarded)
        sA[threadIdx.y][threadIdx.x] =
            (row < M && aCol < K) ? A[row * K + aCol] : 0.0f;

        sB[threadIdx.y][threadIdx.x] =
            (bRow < K && col < N) ? B[bRow * N + col] : 0.0f;

        __syncthreads();

        // Accumulate this tile's contribution to C(row, col)
        for (int k = 0; k < TILE; ++k)
        {
            acc += sA[threadIdx.y][k] * sB[k][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < M && col < N)
    {
        C[row * N + col] = acc;
    }
}

template __global__ void gemm_tiled_kernel<16>(
    const float *__restrict__ A,
    const float *__restrict__ B,
    float *__restrict__ C,
    int M, int N, int K);
template __global__ void gemm_tiled_kernel<32>(
    const float *__restrict__ A,
    const float *__restrict__ B,
    float *__restrict__ C,
    int M, int N, int K);