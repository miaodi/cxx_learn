#include "hip/hip_runtime.h"
#include "gemm.h"

__global__ void gemm_kernel(const float *A, const float *B, float *C, int M,
                            int N, int K)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int row = idx / N;
    int col = idx % N;

    if (row < M && col < N)
    {
        float sum = 0.0f;
        for (int i = 0; i < K; ++i)
        {
            sum += A[row * K + i] * B[i * N + col];
        }
        C[row * N + col] = sum;
    }
}

void gpu_gemm(const float *A, const float *B, float *C, int M, int N, int K)
{
    int totalThreads = M * N;
    int blockSize = 256;
    int numBlocks = (totalThreads + blockSize - 1) / blockSize;

    float *d_A, *d_B, *d_C;
    size_t sizeA = M * K * sizeof(float);
    size_t sizeB = K * N * sizeof(float);
    size_t sizeC = M * N * sizeof(float);
    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);

    hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, sizeC, hipMemcpyHostToDevice);

    gemm_kernel<<<numBlocks, blockSize>>>(d_A, d_B, d_C, M, N, K);
    hipDeviceSynchronize();

    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}